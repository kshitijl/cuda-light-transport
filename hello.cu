#include "hip/hip_runtime.h"
#define GL_GLEXT_PROTOTYPES

#include <GL/gl.h>
#include <GL/glut.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <time.h>

const int width = 1024, height = 768;
GLuint mtexture;

uchar3* h_textureBufferData = nullptr;
uchar3* d_textureBufferData = nullptr;

GLuint gl_pixelBufferObject = 0;
hipGraphicsResource * cudaPboResource = nullptr;

__global__ void ray_trace(uchar3 *output) {
  const int ix = blockIdx.x * blockDim.x + threadIdx.x;
  const int iy = blockIdx.y * blockDim.y + threadIdx.y;

  if(ix < width and iy < height) {
    const int out_idx = width*iy + ix;

    output[out_idx] = uchar3{ix/4,iy/6,10};
  }
}

void render(){
  hipGraphicsMapResources(1, &cudaPboResource, 0);
  size_t num_bytes;
  hipGraphicsResourceGetMappedPointer((void**)&d_textureBufferData,
                                       &num_bytes, cudaPboResource);
   
  dim3 grid_dim{width/32 + (width % 32 > 0), height/32 + (height % 32 > 0)};
  dim3 block_dim{32,32};
  ray_trace<<<grid_dim, block_dim>>>(d_textureBufferData);
 
  hipGraphicsUnmapResources(1, &cudaPboResource, 0);
  
  glClear(GL_COLOR_BUFFER_BIT);

  glEnable(GL_TEXTURE_2D);
  glBindTexture(GL_TEXTURE_2D, mtexture);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_pixelBufferObject);
 
  glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0,
                  width,height,
                  GL_RGB, GL_UNSIGNED_BYTE, 0);
  
  float x = 0.8;
  glBegin(GL_QUADS);
  glTexCoord2f(0.0, 0.0); glVertex3f(-x, -x, 0.5);
  glTexCoord2f(1.0, 0.0); glVertex3f(x, -x, 0.5);
  glTexCoord2f(1.0, 1.0); glVertex3f(x, x, 0.5);
  glTexCoord2f(0.0, 1.0); glVertex3f(-x, x, 0.5);
  glEnd();

  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
  glBindTexture(GL_TEXTURE_2D, 0);
  
  glFlush();
  glutPostRedisplay();  
}

int main(int argc, char **argv) {
  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);
  glutInitWindowSize(width, height);
  glutCreateWindow("Render with CUDA");

  glViewport(0, 0, width, height);
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();

  glMatrixMode(GL_MODELVIEW);
  glLoadIdentity();

  glEnable(GL_TEXTURE_2D);
  glPixelStorei(GL_UNPACK_ALIGNMENT, 1);

  glGenTextures(1, &mtexture);
  glBindTexture(GL_TEXTURE_2D, mtexture);

  glTexImage2D(GL_TEXTURE_2D,
               0,                    // level 0
               3,                    // use only R, G, and B components
               width, height,        // texture has width x height texels
               0,                    // no border
               GL_RGB,               // texels are in RGB format
               GL_UNSIGNED_BYTE,     // color components are unsigned bytes
               h_textureBufferData);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

  glGenBuffers(1, &gl_pixelBufferObject);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_pixelBufferObject);
  glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, width * height * sizeof(uchar3),
               h_textureBufferData, GL_STREAM_COPY);
 
  hipError_t result = hipGraphicsGLRegisterBuffer(&cudaPboResource, gl_pixelBufferObject,
                                                  cudaGraphicsMapFlagsWriteDiscard);

  glutDisplayFunc(render);
  glutMainLoop();
  return 0;
}
